#include "hip/hip_runtime.h"
vecXSwap = [v1 v2] {
	x = v2.x
	v2.x = v1.x
	v1.x = x
}

vecYSwap = [v1 v2] {
	y = v2.y
	v2.y = v1.y
	v1.y = y
}

CRect2 = [
	corner1 = [x=0 y=0]
	corner2 = [x=0 y=0]
	addVec = [v]{
		++( super.corner1.x: v.x: )
		++( super.corner1.y: v.y: )
		++( super.corner2.x: v.x: )
		++( super.corner2.y: v.y: )
	}
	shift = [x y]{
		++( super.corner1.x: x: )
		++( super.corner1.y: y: )
		++( super.corner2.x: x: )
		++( super.corner2.y: y: )
	}
	makeCorner1UpperLeft = {
		if ( gt(super.corner1.x: super.corner2.x:) ) {
			vecXSwap(super.corner1 super.corner2)
		}
		if ( gt(super.corner1.y: super.corner2.y:) ) {
			vecYSwap(super.corner1 super.corner2)
		}
	}
	width = {
		ret(abs(-(corner1.x: corner2.x:)))
	}
	height = {
		ret(abs(-(corner1.y: corner2.y:)))
	}
	area = {
		ret(*(super.width: super.height:))
	}
]
