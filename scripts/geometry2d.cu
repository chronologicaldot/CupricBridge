#include "hip/hip_runtime.h"
makeVec2 = [x y] {ret([x=x y=y])}
makeRect = [x y x2 y2]{ ret([x=x y=y x2=x2 y2=y2]) }
makeCRect = [x y x2 y2]{ ret([corner1=makeVec2(x y) corner2=makeVec2(x2 y2)]) }
makeRectFromVec2 = [v] { ret([x=0 y=0 x2=v.x y2=v.y]) }

vecXSwap = [v1 v2] {
	x = v2.x
	v2.x = v1.x
	v1.x = x
}

vecYSwap = [v1 v2] {
	y = v2.y
	v2.y = v1.y
	v1.y = y
}

Vec2Ops = [
	addVecs = [v1 v2]{
		ret(makeVec2(+(v1.x: v2.x:), +(v1.y: v2.y:)))
	}
	subtractVecs = [v1 v2]{
		ret(makeVec2(-(v1.x: v2.x:), -(v1.y: v2.y:)))
	}
	addScalar = [v s] {
		ret(makeVec2(+(v.x: s:) +(v.y: s:)))
	}
	subtractScalar = [v s] {
		ret(makeVec2(-(v.x: s:) -(v.y: s:)))
	}
	addShiftXY = [v x y]{
		ret(makeVec2(+(v.x: x:) +(v.y: y:)))
	}
	multiplyScalar = [v s]{
		ret(makeVec2(*(v.x: s:), *(v.y: s:)))
	}
	divideScalar = [v s]{
		ret(makeVec2(/(v.x: s:), /(v.y: s:)))
	}
	lengthSquared = [v] {
		ret(+(*(v.x: v.x:) *(v.y: v.y:)))
	}
	dot = [v1 v2]{
		ret(+(*(v1.x: v2.x:) *(v1.y: v2.y:)))
	}
	cross = [v1 v2]{
		ret(-(*(v1.x: v2.y:) *(v1.y: v2.x:)))
	}
]

RectOps = [
	width = [r]{
		ret(abs(-(r.x: r.x2:)))
	}
	height = [r]{
		ret(abs(-(r.y: r.y2:)))
	}
	area = [r]{
		ret(abs(*(-(r.x: r.x2:) -(r.y: r.y2:))))
	}
	addVec = [r v]{
		ret(makeRect(+(r.x: v.x:) +(r.y: v.y:) +(r.x2: v.x:) +(r.y2: v.y:)))
	}
	shiftByVec2 = [r v]{
		++( r.x: v.x: )
		++( r.y: v.y: )
		++( r.x2: v.x: )
		++( r.y2: v.y: )
	}
	makeSameSizeRect = [r]{
		ret(makeRect(0 0 RectOps.width(r) RectOps.height(r)))
	}
]

makeVec2FromRect = [r]{
	ret( makeVec2(RectOps.width(r) RectOps.height(r)) )
}
